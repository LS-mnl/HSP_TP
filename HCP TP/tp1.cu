
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
*** Function Name : MatrixInit ***


 Initializes a matrix of size NxP with random values between -1 and 1.

 * Parameters:
 *    M: Pointer to the matrix (stored in row-major order)
 *    n: Number of rows in the matrix
 *    p: Number of columns in the matrix
*/


void MatrixInit(float *M, int n, int p) {
    for (int i = 0; i < n * p; i++) {
        M[i] = ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
    }
}

/*
*** Function Name : MatrixPrint ***

Sert à afficher une matrice NxP dans une forme plus conventionnelle. 

                                                              0 0 0
ex : M = [0 0 0; 0 0 0; 0 0 0] sera affichée comme suit : M = 0 0 0   
                                                              0 0 0 

Paramètres : 
    n : nombre de lignes de la matrice,
    p : nombre de colonnes de la matrice si n différent de p,
    M : pointeur de la matrice
*/

void MatrixPrint(float *M, int n, int p) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < p; j++) {
            printf("%1.1f ", M[i * p + j]);
        }
        printf("\n");
    }
    printf("\n");
}

/*
*** Function Name : MatrixAdd ***

Sert à additionner deux matrices de même taille NxP sur le CPU

Paramètres : 
    n : nombre de lignes des matrice,
    p : nombre de colonnes des matrice si n différent de p,
    M1 : pointeur de la matrice 1 de taille NxP,
    M2 : pointeur de la matrice 2 de taille NxP,
    Mout : pointeur vers la matrice résultante de l'addition de taille NxP
*/

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    for (int i = 0; i < n * p; i++) {
        Mout[i] = M1[i] + M2[i];
    }
}
/*
*** Function Name : cudaMatrixAdd ***

Sert à additionner deux matrices de même taille NxP sur le GPU 

Paramètres : 
    n : nombre de lignes des matrice,
    p : nombre de colonnes des matrices si n différent de p,
    M1 : pointeur de la matrice 1 de taille NxP,
    M2 : pointeur de la matrice 2 de taille NxP,
    Mout : pointeur vers la matrice résultante de l'addition de taille NxP,
    
On peut considérer les dimensions des matrices comme les paramètres gridDim et blockDim pour l'appel de la fonction:
    les lignes correspondent aux blocks,
    les colonnes correspondent aux threads
*/

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p) {
    int idx = threadIdx.x * blockDim.y + threadIdx.y;
    if (idx < n * p) {
        Mout[idx] = M1[idx] + M2[idx];
    }
}

/*
*** Function Name : MatrixMult ***

Sert à effectuer la multiplication matricielle (dot) de deux matrices carrées NxN sur CPU

Paramètres : 
    n : nombre de lignes et colonne des matrices,
    M1 : pointeur de la matrice 1 de taille NxN,
    M2 : pointeur de la matrice 2 de taille NxN,
    Mout : pointeur vers la matrice résultante de l'addition de taille NxN,
*/

void MatrixMult(float *M1, float *M2, float *Mout, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            float s = 0.0f;
            for (int k = 0; k < n; k++) {
                s += M1[i * n + k] * M2[k * n + j];
            }
            Mout[i * n + j] = s;
        }
    }
}

/*
*** Function Name : cudaMatrixMult ***

Sert à effectuer la multiplication matricielle (dot) de deux matrices carrées NxN sur GPU

Paramètres : 
    n : nombre de lignes et de colonnes des matrices,
    M1 : pointeur de la matrice 1 de taille NxN,
    M2 : pointeur de la matrice 2 de taille NxN,
    Mout : pointeur vers la matrice résultante de la multiplication de taille NxN,
    
On peut considérer les dimensions des matrices comme les paramètres gridDim et blockDim pour l'appel de la fonction:
    les lignes correspondent aux blocks,
    les colonnes correspondent aux threads
*/

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float val = 0;
        for (int k = 0; k < n; ++k) {
            val += M1[row * n + k] * M2[k * n + col];
        }
        Mout[row * n + col] = val;
    }
}
/*
*** Function Name : cudaMatrixMultGeneral ***

Sert à effectuer la multiplication matricielle (dot) d'une matrice NxP avec une matrice PxM sur le GPU

Paramètres : 
    n : nombre de lignes de la matrice M1
    p : nombre de colonnes de M1, de lignes de M2
    m : nombre de colonnes de M2
    M1 : pointeur de la matrice 1 de taille NxP,
    M2 : pointeur de la matrice 2 de taille PxM,
    Mout : pointeur vers la matrice résultante de la multiplication de taille NxM

On peut considérer les dimensions de la matrice de sortie comme les paramètres gridDim et blockDim pour l'appel de la fonction:
    les lignes correspondent aux blocks : n
    les colonnes correspondent aux threads : m
*/

__global__ void cudaMatrixMultGeneral(float *M1, float *M2, float *Mout, int n, int p, int m) {
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float s = 0.0f;

    if (lig < n && col < m) {
        for (int i = 0; i < p; i++) {
            s += M1[lig * p + i] * M2[i * m + col];
        }
        Mout[lig * m + col] = s;
    }
}









int main() {

    // CPU \\

    // Test de MatrixInit et MatrixPrint
    float *M;

    int n = 3;
    int p = 3;
    int m = 3;

    // Allocation de la mémoire pour la création de la matrice
    M = (float *)malloc(n * p * sizeof(float));
    MatrixInit(M, n, p);

    free(M);

    // Test de MatrixAdd
    float *M1, *M2, *Mout;

    // Allocation des mémoires
    M1 = (float *)malloc(n * p * sizeof(float));
    M2 = (float *)malloc(p * m * sizeof(float));
    Mout = (float *)malloc(n * m * sizeof(float));

    MatrixInit(M1, n, p);
    MatrixInit(M2, p, m);

    // Test de MatrixAdd et MatrixMult sur CPU
    MatrixAdd(M1, M2, Mout, n, p);

    printf("\nMatrix 1\n");
    MatrixPrint(M1, n, p);
    printf("Matrix 2\n");
    MatrixPrint(M2, n, p);
    printf("Result Matrix from Addition on CPU:\n");
    MatrixPrint(Mout, n, p);

    MatrixMult(M1, M2, Mout, n);

    printf("\nMatrix 1\n");
    MatrixPrint(M1, n, p);
    printf("Matrix 2\n");
    MatrixPrint(M2, n, p);
    printf("Result Matrix from Multiplication on CPU:\n");
    MatrixPrint(Mout, n, p);

    // Test de cudaMatrixAdd
    float *d_M1, *d_M2, *d_Mout;

    // Allocation des mémoires des matrices pour cuda
    hipMalloc((void **)&d_M1, sizeof(float) * n * p);
    hipMalloc((void **)&d_M2, sizeof(float) * p * m);
    hipMalloc((void **)&d_Mout, sizeof(float) * n * m);

    hipMemcpy(d_M1, M1, sizeof(float) * n * p, hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2, sizeof(float) * p * m, hipMemcpyHostToDevice);


    // GPU \\

    dim3 block_size(n, m);
    dim3 grid_size(1, 1);

    // Addition sur GPU
    cudaMatrixAdd<<<grid_size, block_size>>>(d_M1, d_M2, d_Mout, n, p);
    hipDeviceSynchronize();

    // Copie du résultat sur CPU
    hipMemcpy(Mout, d_Mout, sizeof(float) * n * m, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("\nMatrix 1\n");
    MatrixPrint(M1, n, p);
    printf("Matrix 2\n");
    MatrixPrint(M2, p, m);
    printf("Result Matrix from Addition on GPU:\n");
    MatrixPrint(Mout, n, m);


    // Multiplication sur GPU 
    cudaMatrixMultGeneral<<<grid_size, block_size>>>(d_M1, d_M2, d_Mout, n, p, m);
    hipDeviceSynchronize();

    // Copie du résultat sur CPU

    hipMemcpy(Mout, d_Mout, sizeof(float) * n * m, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("\nMatrix 1\n");
    MatrixPrint(M1, n, p);
    printf("Matrix 2\n");
    MatrixPrint(M2, p, m);
    printf("Result Matrix from Multiplication on GPU:\n");
    MatrixPrint(Mout, n, m);

    hipFree(d_M1);
    hipFree(d_M2);
    hipFree(d_Mout);

    free(M1);
    free(M2);
    free(Mout);

    return 0;
}

    
