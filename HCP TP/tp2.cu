
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
*** Function Name : MatrixInit ***

Initializes a matrix of size NxPxD with specified values based on the 'type':
   type == 0: Fills the matrix with zeros.
   type == 1: Fills the matrix with ones.
   type == 2: Creates a kernel matrix with a central value of 2, others being zero (assumes a 3D matrix).
   type == 3: Fills the matrix with random values between 0 and 1.
Parameters:
   M: Pointer to the matrix (in row-major order).
   n: Number of rows.
   p: Number of columns.
   d: Depth (third dimension).
   type: Initialization mode (0, 1, 2, or 3).
*/


void MatrixInit(float *M, int n, int p, int d, int type){
    
    float random_value;
    
    if (type == 0){
        for (int i = 0; i < n * p * d; i++){
            M[i] =  0;
        }
    }
    if (type == 1){
        for (int i = 0; i < n * p * d; i++){
            M[i] =  1;
        }
    }
    else if (type == 2){
        for (int i = 0; i < n * p * d; i++){
            M[i] =  0;
        }
        for (int k = 0; k < d; k++){
            M[k * (n * p) + 12] = 2;
        }
    }
    else{
        //Valeurs entre 0 et 1
        for (int i = 0; i < n * p * d; i++){
            random_value = (float)rand() / (float)(RAND_MAX/1.0);
            M[i] =  random_value;
        }
    }
}

/*
*** Function Name : MatrixPrint2D ***
Prints a 2D matrix in conventional format.
Parameters:
   M: Pointer to the matrix (in row-major order).
   n: Number of rows.
   p: Number of columns.
*/

void MatrixPrint2D(float *M, int n, int p){
    
    printf("\n");
    for (int lig = 0; lig < p; lig++){
        for(int col = lig * n; col < n * (lig+1); col++){
            printf("%1.1f ", M[col]);
        }
        printf("\n");
    }
    printf("\n");
}

// Layer 2 - Convolution 2D

/*
*** Function Name : cudaConv2D ***

Performs 2D convolution on a matrix M using a specified number of 5x5 kernels.

Parameters:
   M: Input matrix in device memory.
   kernel: Convolution kernels in device memory.
   Mout: Output matrix in device memory to store the result.
   M_ligne: Number of rows in the input matrix.
   M_colonne: Number of columns in the input matrix.
   kernel_size: Size of one side of the square convolution kernel.
   nb_kernel: Number of convolution kernels.
   Mout_ligne: Number of rows in the output matrix.
   Mout_colonne: Number of columns in the output matrix.

Note:
   The dimensions of the output matrix Mout are computed as:
   Mout_ligne = (M_ligne - kernel_size) + 1
   Mout_colonne = (M_colonne - kernel_size) + 1
*/


__global__ void cudaConv2D(float* M, float* kernel, float* Mout, int M_ligne, int M_colonne, int kernel_size, int nb_kernel, int Mout_ligne, int Mout_colonne){
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float s;

    if (lig < Mout_ligne && col < Mout_colonne){
        
        int tot_kernel = kernel_size * kernel_size;
        int tot_Mout = Mout_ligne * Mout_colonne;
        
        for (int n_k = 0; n_k < nb_kernel; n_k++){
            s = 0.0;
            
            for (int kernel_lig = 0; kernel_lig < kernel_size; kernel_lig++) {
                for (int kernel_col = 0; kernel_col < kernel_size; kernel_col++) {
                    
                    s += M[(lig + kernel_lig) * M_colonne + (col + kernel_col)] * kernel[kernel_lig * kernel_size + kernel_col + n_k * tot_kernel];
                    
                }
            }
            
            Mout[lig * Mout_colonne + col + n_k * tot_Mout] = s;
        }
    }
}

// Layer 3 - Sous-échantillonnage 


/*
*** Function Name : cudaMeanPool ***

Performs mean pooling on the input matrix M using a 2x2 kernel.

Example:
    Given a sub-matrix:   1 2
                      	  3 4
    The mean pool result is: (1 + 2 + 3 + 4) / 4 = 2.5

Parameters:
    M: Pointer to the input matrix.
    Mout: Pointer to the output matrix.
    M_ligne: Number of rows in the input matrix M.
    M_colonne: Number of columns in the input matrix M.
    M_prof: Depth of the input matrix M.
    meanpool_size: The size of the mean pooling window (both rows and columns).
    Mout_ligne: Number of rows in the output matrix Mout.
    Mout_colonne: Number of columns in the output matrix Mout.

Note:
    The relationship between the input and output dimensions is:
    Mout_ligne = M_ligne / meanpool_size
    Mout_colonne = M_colonne / meanpool_size
*/



__global__ void cudaMeanPool(float* M, float* Mout, int M_ligne, int M_colonne, int M_prof, int meanpool_size, int Mout_ligne, int Mout_colonne){
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (lig % meanpool_size == 0 && col % meanpool_size == 0){
        
        float s;
        int tot_meanpool = meanpool_size * meanpool_size;
        int tot_M = M_ligne * M_colonne;
        int tot_Mout = Mout_ligne * Mout_colonne;
        
        for (int n_prof = 0; n_prof < M_prof; n_prof++){
            s = 0.0;
            
            for (int meanpool_lig = 0; meanpool_lig < meanpool_size; meanpool_lig++) {
                for (int meanpool_col = 0; meanpool_col < meanpool_size; meanpool_col++) {
                    s += M[(lig + meanpool_lig) * M_colonne + col + meanpool_col + n_prof * tot_M] / tot_meanpool;
            
                }
            }
            if (lig == 0){
                Mout[lig * Mout_colonne + (col / meanpool_size) + n_prof * tot_Mout] = s;
            }
            else if (col == 0){
                Mout[(lig / meanpool_size) * Mout_colonne + col + n_prof * tot_Mout] = s;
            }
            else{
                Mout[(lig / meanpool_size) * Mout_colonne + (col / meanpool_size) + n_prof * tot_Mout] = s;
            }
        }
    }
}

/*
*** Function Name : activation_tanh ***

Applies the hyperbolic tangent (tanh) activation function to each element of matrix M on the GPU.

Note: This is a __device__ function and must be called from a __global__ function on the GPU.

Parameters:
    M: Pointer to the matrix on which to apply the tanh function.
    M_ligne: Number of rows in the matrix M.
    M_colonne: Number of columns in the matrix M.
    M_prof: Depth of the matrix M.
*/


__device__ float* activation_tanh(float* M, int M_ligne, int M_colonne, int M_prof){
    
    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (lig < M_ligne && col < M_colonne){
        
        int tot_M = M_ligne * M_colonne;
        
        for (int n_prof = 0; n_prof < M_prof; n_prof++){
            M[lig * M_colonne + col + n_prof * tot_M] = tanh(M[lig * M_colonne + col + n_prof * tot_M]);
        }
            
    }
            
    return M;
}

/*
Kernel function to call the activation_tanh __device__ function.

Parameters:
    M: Pointer to the matrix on the GPU.
    M_ligne: Number of rows in the matrix M.
    M_colonne: Number of columns in the matrix M.
    M_prof: Depth of the matrix M.
*/


__global__ void cudaTanh(float* M, int M_ligne, int M_colonne, int M_prof){
    activation_tanh(M, M_ligne, M_colonne, M_prof);
}

/*
*** Function Name : cudaTanh ***

/*
The cudaTanh kernel launches the device-level function activation_tanh across GPU threads.

This kernel function serves as a wrapper to invoke the activation_tanh function, which 
applies the hyperbolic tangent (tanh) activation function to each element of the input matrix M.

Parameters:
   M: Pointer to the input matrix in device memory.
   M_ligne: The number of rows in the input matrix.
   M_colonne: The number of columns in the input matrix.
   M_prof: The depth of the input matrix, indicating the number of matrices in the case of a 3D matrix.

Note: 
   This kernel should be configured with an appropriate number of blocks and threads to match the size of the input matrix.
   It is assumed that the input matrix M is stored in a flat, row-major format.
*/



int main(){
    
  // CPU \\ 
    
    
    // Création de l'image d'entrée à convoluer
    float *raw_data;    
    raw_data = (float*)malloc(32 * 32 * 1 * sizeof(float));
    
    MatrixInit(raw_data, 32, 32, 1, 1);
    
    // Création de la sortie de la conv2D
    float *C1_data;    
    C1_data = (float*)malloc(28 * 28 * 6 * sizeof(float));
    
    MatrixInit(C1_data, 28, 28, 6, 0);
    
    // Création de la sortie du sous-échantillonnage
    float *S1_data;    
    S1_data = (float*)malloc(14 * 14 * 6 * sizeof(float));
    
    MatrixInit(S1_data, 14, 14, 6, 0);
    
    
    // Création des premiers noyaux de convolution
    float *C1_kernel;    
    C1_kernel = (float*)malloc(5 * 5 * 6 * sizeof(float));
    
    MatrixInit(C1_kernel, 5, 5, 6, 2);

    
// GPU \\ 

    // Définition des matrices cuda
    float *d_raw_data, *d_C1_data, *d_C1_kernel, *d_S1_data;
    
    // Allocation des mémoires des matrices pour cuda
    hipMalloc((void**)&d_raw_data, sizeof(float) * 32 * 32 * 1);
    hipMalloc((void**)&d_C1_kernel, sizeof(float) * 5 * 5 * 6);
    hipMalloc((void**)&d_C1_data, sizeof(float) * 28 * 28 * 6);
    hipMalloc((void**)&d_S1_data, sizeof(float) * 14 * 14 * 6);
    
    // Copie des valeurs des matrices initialisées sur le CPU dans leur homonyme GPU
    hipMemcpy(d_raw_data, raw_data, sizeof(float) * 32 * 32 * 1, hipMemcpyHostToDevice);
    hipMemcpy(d_C1_kernel, C1_kernel, sizeof(float) * 5 * 5 * 6, hipMemcpyHostToDevice);
    hipMemcpy(d_C1_data, C1_data, sizeof(float) * 28 * 28 * 6, hipMemcpyHostToDevice);
    hipMemcpy(d_S1_data, S1_data, sizeof(float) * 14 * 14 * 6, hipMemcpyHostToDevice);
  
// GPU \\ 

    // Process sur GPU
    dim3 block_size(32, 32);
    dim3 grid_size(1,1);
    
    cudaConv2D<<<grid_size, block_size>>>(d_raw_data, d_C1_kernel, d_C1_data, 32, 32, 5, 6, 28, 28);
    hipDeviceSynchronize();
    
    cudaTanh<<<grid_size, block_size>>>(d_C1_data, 28, 28, 6);
    hipDeviceSynchronize();
    
    cudaMeanPool<<<grid_size, block_size>>>(d_C1_data, d_S1_data, 28, 28, 6, 2, 14, 14);
    hipDeviceSynchronize();
    
    
    // Copie des résultats sur CPU
    hipMemcpy(C1_data, d_C1_data, sizeof(float) * 28 * 28 * 6, hipMemcpyDeviceToHost);
    hipMemcpy(S1_data, d_S1_data, sizeof(float) * 14 * 14 * 6, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    printf("\nMatrice de base raw_data:");
    MatrixPrint2D(raw_data, 32, 32);
    printf("Noyau de convolution C1_kernel:");
    MatrixPrint2D(C1_kernel, 5, 5);
    printf("Matrice résultante de la convolution et de la fonction d'activation:");
    MatrixPrint2D(C1_data, 28, 28);
    printf("Matrice résultante du MeanPooling:");
    MatrixPrint2D(S1_data, 14, 14);
    
    hipFree(d_raw_data);
    hipFree(d_C1_kernel);
    hipFree(d_C1_data);
    hipFree(d_S1_data);
    
    free(raw_data);
    free(C1_data);
    free(S1_data);
    free(C1_kernel);
}
